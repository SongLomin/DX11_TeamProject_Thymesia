#include "hip/hip_runtime.h"
#include "Update_Particle.cuh"


__device__  PARTICLE_DESC Play_TestInternal(const _uint& i, _float fTimeDelta, _matrix BoneMatrix, PARTICLE_DESC* input)
{
	PARTICLE_DESC desc = *input;
	desc.fCurrentLifeTime = 999.f;

	return desc;
}


__global__ void kernel_UpdateParticle(_float fTimeDelta, _matrix BoneMatrix, PARTICLE_DESC* input, PARTICLE_DESC* output, _int size)
{
	_int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < size)
	{
		*output = Play_TestInternal(i, fTimeDelta, BoneMatrix, input);
	}


}

__global__ void Kernel(int size)
{

}

void CudaMain_UpdateParticle(float fTimeDelta, _matrix BoneMatrix, PARTICLE_DESC* input, PARTICLE_DESC* output, _int size)
{

}

